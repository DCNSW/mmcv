// Modified from
// https://github.com/sshaoshuai/Pointnet2.PyTorch/tree/master/pointnet2/src/ball_query_gpu.cu

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "ball_query_cuda_kernel.cuh"
#include "pytorch_cuda_helper.hpp"

#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

void ball_query_kernel_launcher(int b, int n, int m, float min_radius,
                                float max_radius, int nsample,
                                const float *new_xyz, const float *xyz,
                                int *idx) {
  // new_xyz: (B, M, 3)
  // xyz: (B, N, 3)
  // output:
  //      idx: (B, M, nsample)

  hipError_t err;

  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

  dim3 blocks(DIVUP(m, THREADS_PER_BLOCK),
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  ball_query_cuda_kernel<<<blocks, threads, 0, stream>>>(
      b, n, m, min_radius, max_radius, nsample, new_xyz, xyz, idx);
  // hipDeviceSynchronize();  // for using printf in kernel function
  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}
